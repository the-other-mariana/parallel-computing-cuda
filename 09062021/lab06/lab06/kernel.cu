#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h> /* srand, rand */
#include <time.h> /* time */

#include<iostream>
using namespace std;
__global__ void idKernel(int* vecA, int* vecB, int* vecC) {
	int gId = threadIdx.x + blockDim.x * blockIdx.x;

	vecA[gId] = threadIdx.x;
	vecB[gId] = blockIdx.x;
	vecC[gId] = gId;
}

void printArray(int* arr, int size, char * msg) {
	cout << msg << ": ";
	for (int i = 0; i < size; i++) {
		printf("%d ", arr[i]);
	}
	printf("\n");
}

int main()
{
	const int vectorSize = 64;
	int* host_a = (int*)malloc(sizeof(int) * vectorSize);
	int* host_b = (int*)malloc(sizeof(int) * vectorSize);
	int* host_c = (int*)malloc(sizeof(int) * vectorSize);

	int* dev_a, * dev_b, * dev_c;

	hipMalloc((void**)&dev_a, sizeof(int) * vectorSize);
	hipMalloc((void**)&dev_b, sizeof(int) * vectorSize);
	hipMalloc((void**)&dev_c, sizeof(int) * vectorSize);

	srand(time(NULL));

	for (int i = 0; i < vectorSize; i++) {
		host_a[i] = 0;
		host_b[i] = 0;
		host_c[i] = 0;
	}

	hipMemcpy(dev_a, host_a, sizeof(int) * vectorSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, sizeof(int) * vectorSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_c, host_c, sizeof(int) * vectorSize, hipMemcpyHostToDevice);

	dim3 grid(1, 1, 1);
	dim3 block(64, 1, 1);
	idKernel << < grid, block >> > (dev_a, dev_b, dev_c);
	hipDeviceSynchronize(); // wait until kernel finishes and then come back to following code
	hipMemcpy(host_a, dev_a, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);
	hipMemcpy(host_b, dev_b, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);
	hipMemcpy(host_c, dev_c, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);

	printf("Execution 1: 1 block 64 threads \n");
	printArray(host_a, vectorSize, "threadIdx.x");
	printArray(host_b, vectorSize, "blockIdx.x");
	printArray(host_c, vectorSize, "globalId");

	grid.x = 64; // (64, 1, 1)
	block.x = 1; // (1, 1, 1)
	idKernel << < grid, block >> > (dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	hipMemcpy(host_a, dev_a, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);
	hipMemcpy(host_b, dev_b, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);
	hipMemcpy(host_c, dev_c, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);

	printf("\nExecution 2: 64 blocks 1 thread \n");
	printArray(host_a, vectorSize, "threadIdx.x");
	printArray(host_b, vectorSize, "blockIdx.x");
	printArray(host_c, vectorSize, "globalId");

	grid.x = 4;
	block.x = 16;
	idKernel << < grid, block >> > (dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	hipMemcpy(host_a, dev_a, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);
	hipMemcpy(host_b, dev_b, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);
	hipMemcpy(host_c, dev_c, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);

	printf("\nExecution 3: 4 block 16 threads \n");
	printArray(host_a, vectorSize, "threadIdx.x");
	printArray(host_b, vectorSize, "blockIdx.x");
	printArray(host_c, vectorSize, "globalId");

	free(host_a);
	free(host_b);
	free(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}