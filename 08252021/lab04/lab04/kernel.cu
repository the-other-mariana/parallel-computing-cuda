﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void kernel()
{
    printf("threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d | blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d | blockDim.x: %d, blockDim.y: %d, blockDim.z: %d | gridDim.x: %d, gridDim.y: %d, gridDim.z: %d\n", 
        threadIdx.x, threadIdx.y, threadIdx.z, 
        blockIdx.x, blockIdx.y, blockIdx.z,
        blockDim.x, blockDim.y, blockDim.z,
        gridDim.x, gridDim.y, gridDim.z);
}

int main() {
    dim3 grid(2, 2, 2);
    dim3 block(2, 2, 2);
    kernel <<< grid, block >>> ();

    return 0;
}