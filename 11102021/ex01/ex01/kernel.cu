#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define N 3
__constant__ int dev_A[N * N];

using namespace std;

__host__ void checkCUDAError(const char* msg) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), msg);
	}
}

__host__ void validate(int* result_CPU, int* result_GPU, int size) {
	if (*result_CPU != *result_GPU) {
		printf("The results are not equal.\n");
		return;
	}
	printf("Kernel validated successfully.\n");
	return;
}

__host__ void CPU_fn(int* v, int* sum, const int size) {
	for (int i = 0; i < size; i++) {
		*sum += v[i];
	}
}

__global__ void kernel(int* res) {
	int gId = threadIdx.x + (blockDim.x * threadIdx.y);
	res[gId] = dev_A[N * threadIdx.x + threadIdx.y];
}

__host__ void printMtx(int* mtx) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << mtx[(i * N) + j] << " ";
		}
		cout << endl;
	}
}

int main() {

	int* dev_B;
	int* host_B = (int*)malloc(sizeof(int) * N * N);
	int* host_A = (int*)malloc(sizeof(int) * N * N);
	hipMalloc((void**)&dev_B, sizeof(int) * N * N);

	for (int i = 0; i < N * N; i++) {
		host_A[i] = i + 1;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(dev_A), host_A, sizeof(int) * N * N);

	dim3 grid(1);
	dim3 block(N, N);
	kernel << < grid, block >> > (dev_B);
	hipMemcpy(host_B, dev_B, sizeof(int) * N * N, hipMemcpyDeviceToHost);

	printMtx(host_A);
	cout << endl;
	printMtx(host_B);

	return 0;
}