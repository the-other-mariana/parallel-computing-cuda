#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define N 32
__constant__ int dev_A[N * N];

using namespace std;

__host__ void checkCUDAError(const char* msg) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), msg);
	}
}

__host__ void validate(int* result_CPU, int* result_GPU) {
	for (int i = 0; i < N * N; i++) {
		if (*result_CPU != *result_GPU) {
			printf("[FAILED] The results are not equal.\n");
			return;
		}
	}
	printf("[SUCCESS] Kernel validation.\n");
	return;
}

__host__ void CPU_transpose(int* vector, int* res) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			res[(i * N) + j] = vector[(N * j) + i];
		}
	}
}

__global__ void GPU_transpose(int* res) {
	int gId = threadIdx.x + (blockDim.x * threadIdx.y);
	res[gId] = dev_A[N * threadIdx.x + threadIdx.y];
}

__host__ void printMtx(int* mtx) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << mtx[(i * N) + j] << " ";
		}
		cout << endl;
	}
}

int main() {

	int* dev_B;
	int* host_B = (int*)malloc(sizeof(int) * N * N);
	int* cpu_B = (int*)malloc(sizeof(int) * N * N);
	int* host_A = (int*)malloc(sizeof(int) * N * N);

	hipMalloc((void**)&dev_B, sizeof(int) * N * N);
	checkCUDAError("Error at hipMalloc: dev_B");

	for (int i = 0; i < N * N; i++) {
		host_A[i] = i + 1;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(dev_A), host_A, sizeof(int) * N * N);
	checkCUDAError("Error at MemcpyToSymbol");

	dim3 grid(1);
	dim3 block(N, N);
	GPU_transpose << < grid, block >> > (dev_B);
	checkCUDAError("Error at kernel");
	hipMemcpy(host_B, dev_B, sizeof(int) * N * N, hipMemcpyDeviceToHost);
	checkCUDAError("Error at Memcpy host_B <- dev_B");

	CPU_transpose(host_A, cpu_B);

	printf("Input: \n");
	printMtx(host_A);
	printf("CPU: \n");
	printMtx(cpu_B);
	printf("GPU: \n");
	printMtx(host_B);

	validate(cpu_B, host_B);

	free(host_B);
	free(cpu_B);
	free(host_A);
	hipFree(dev_B);

	return 0;
}