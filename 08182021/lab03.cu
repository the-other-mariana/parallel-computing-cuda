#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void solveGPU(double* dev_abc, double* dev_x1x2, bool* dev_error)
{
    double root = (dev_abc[1] * dev_abc[1]) - (4 * dev_abc[0] * dev_abc[2]);
    // printf("root: %lf\n", root);
    if (root < 0) {
        *dev_error = true;
    }
    else {
        *dev_error = false;
        dev_x1x2[0] = ((-1 * dev_abc[1] - sqrt(root)) / (2 * dev_abc[0]));
        dev_x1x2[1] = ((-1 * dev_abc[1] + sqrt(root)) / (2 * dev_abc[0]));
    }
     
}

int main() {
    double* n_host = (double*)malloc(sizeof(double) * 3);
    double* x1x2_host = (double*)malloc(sizeof(double) * 2);
    bool* error_host = (bool*)malloc(sizeof(bool));

    double* n_dev;
    double* x1x2_dev;
    bool* error_dev;
    hipMalloc((void**)&n_dev, sizeof(double) * 3);
    hipMalloc((void**)&x1x2_dev, sizeof(double) * 2);
    hipMalloc((void**)&error_dev, sizeof(bool));

    for (int i = 0; i < 3; i++) {
        scanf("%lf", &n_host[i]);
    }

    x1x2_host[0] = 0;
    x1x2_host[1] = 0;
    *error_host = false;

    hipMemcpy(n_dev, n_host, sizeof(double) * 3, hipMemcpyHostToDevice);
    hipMemcpy(x1x2_dev, x1x2_host, sizeof(double) * 2, hipMemcpyHostToDevice);
    hipMemcpy(error_dev, error_host, sizeof(bool), hipMemcpyHostToDevice);
    
    solveGPU <<< 1, 1 >>> (n_dev, x1x2_dev, error_dev);

    hipMemcpy(error_host, error_dev, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(x1x2_host, x1x2_dev, sizeof(double) * 2, hipMemcpyDeviceToHost);
    if (*error_host) {
        printf("GPU Result:\n");
        printf("The solution does not exist\n");
    }
    else {
        printf("GPU Result:\n");
        printf("x1 = %lf x2 = %lf\n", x1x2_host[0], x1x2_host[1]);
    }
    
}