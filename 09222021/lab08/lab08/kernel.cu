#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__host__ void checkCUDAError(const char* msg) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), msg);
	}
}

__global__ void kernel(int* m, int* r) {
	int gId = threadIdx.x + threadIdx.y * blockDim.x;
	int n1 = gId - 1;
	int n2 = gId + 1;
	int n3 = gId - blockDim.x;
	int n4 = gId + blockDim.x;
	if (threadIdx.x == 0 || threadIdx.x == (blockDim.x - 1) || threadIdx.y == 0 || threadIdx.y == (blockDim.y - 1)) {
		r[gId] = m[gId];
	}
	else {
		int avg = (m[n1] + m[n2] + m[n3] + m[n4]) / 4;

		r[gId] = avg;
	}
}

int main() {
	const int size = 6;

	int m[size][size] = { 0 };
	int r[size][size] = { 0 };
	int m_vec[size * size] = { 0 };
	int r_vec[size * size] = { 0 };

	int* dev_m, * dev_r;
	hipMalloc((void**)&dev_m, sizeof(int) * size * size);
	checkCUDAError("Error at hipMalloc for dev_m");
	hipMalloc((void**)&dev_r, sizeof(int) * size * size);
	checkCUDAError("Error at hipMalloc for dev_r");

	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			m[i][j] = (int)(rand() % 10);
			m_vec[j + i * size] = m[i][j];
		}
	}

	printf("Original Matrix:\n");
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			printf("%d ", m[i][j]);
		}
		printf("\n");
	}

	for (int i = 0; i < size * size; i++) {
		//printf("%d ", m_vec[i]);
	}
	printf("\n");

	hipMemcpy(dev_m, m_vec, sizeof(int) * size * size, hipMemcpyHostToDevice);
	checkCUDAError("Error at hipMemcpy Host -> Device");

	dim3 grid(1);
	dim3 block(size, size);
	kernel << < grid, block >> > (dev_m, dev_r);
	checkCUDAError("Error at kernel");

	hipMemcpy(r_vec, dev_r, sizeof(int) * size * size, hipMemcpyDeviceToHost);
	checkCUDAError("Error at hipMemcpy Device -> Host");

	printf("Average Matrix:\n");
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			r[i][j] = r_vec[j + i * size];
			printf("%d ", r[i][j]);
		}
		printf("\n");
	}

	hipFree(dev_m);
	hipFree(dev_r);
}