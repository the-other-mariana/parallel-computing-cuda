﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__host__ void checkCUDAError(const char* msg) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), msg);
	}
}

__host__ void validate(int* result_CPU, int* result_GPU, int size) {
	if (*result_CPU != *result_GPU) {
		printf("The results are not equal.\n");
		return;
	}
	printf("Kernel validated successfully.\n");
	return;
}

__host__ void CPU_fn(int *v, int* sum, const int size) {
	for (int i = 0; i < size; i++) {
		*sum += v[i];
	}
}

__global__ void kernel(int* v, int* sum) {
	int gId = threadIdx.x;
	int step = blockDim.x;

	while (step) {
		if (gId < step) {
			v[gId] = v[gId] + v[gId + step];
		}
		step = step / 2;
	}
	if (gId == 0) {
		*sum = v[gId];
	}
}

int main() {

	const int size = 1024;
	int* v = (int*)malloc(sizeof(int) * size);
	int sumCPU = 0;
	int sumGPU = 0;

	int* dev_v, *sum;
	hipMalloc((void**)&dev_v, sizeof(int) * size);
	hipMalloc((void**)&sum, sizeof(int));

	for (int i = 0; i < size; i++) {
		v[i] = 1;
	}

	hipMemcpy(dev_v, v, sizeof(int) * size, hipMemcpyHostToDevice);
	hipMemcpy(sum, &sumGPU, sizeof(int), hipMemcpyHostToDevice);

	dim3 grid(1);
	dim3 block(size);
	
	kernel <<< grid, block >>> (dev_v, sum);
	hipMemcpy(&sumGPU, sum, sizeof(int), hipMemcpyDeviceToHost);
	printf("GPU sum: %d\n", sumGPU);

	CPU_fn(v, &sumCPU, size);
	printf("CPU sum: %d\n", sumCPU);

	validate(&sumCPU, &sumGPU, size);

	return 0;
}