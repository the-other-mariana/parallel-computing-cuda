﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__host__ int addCPU(int* num1, int* num2) {
    return(*num1 + *num2);
}

// kernel: __global__
__global__ void addGPU(int* num1, int* num2, int* res)
{
    *res = *num1 + *num2;
}

int main()
{
    // reserve mem in host
    int* host_num1 = (int*)malloc(sizeof(int)); // could be a simple integer and then you pass as param the &variable
    int* host_num2 = (int*)malloc(sizeof(int));
    int* host_resCPU = (int*)malloc(sizeof(int));
    int* host_resGPU = (int*)malloc(sizeof(int));

    // reserve mem in dev
    int* dev_num1, * dev_num2, * dev_res;
    hipMalloc((void**)&dev_num1, sizeof(int));
    hipMalloc((void**)&dev_num2, sizeof(int));
    hipMalloc((void**)&dev_res, sizeof(int)); // this pointer points to an address in the device

    // init data
    *host_num1 = 2;
    *host_num2 = 3;
    *host_resCPU = 0;
    *host_resGPU = 0;

    // data transfer
    hipMemcpy(dev_num1, host_num1, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_num2, host_num2, sizeof(int), hipMemcpyHostToDevice);

    // CPU call to CPU func
    *host_resCPU = addCPU(host_num1, host_num2);
    printf("CPU result \n");
    printf("%d + %d = %d \n", *host_num1, *host_num2, *host_resCPU);

    // CPU call to GPU func
    addGPU <<< 1, 1 >>> (dev_num1, dev_num2, dev_res);
    // dev_res is a pointer made with cudaMalloc (Global Memory)
    hipMemcpy(host_resGPU, dev_res, sizeof(int), hipMemcpyDeviceToHost);
    printf("GPU result \n");
    // dev_num1 is án address in GPU, you cannot access it from CPU
    printf("%d + %d = %d \n", *host_num1, *host_num2, *host_resGPU);

    // free memory
    free(host_num1);
    free(host_num2);
    free(host_resCPU);
    free(host_resGPU);

    hipFree(dev_num1);
    hipFree(dev_num2);
    hipFree(dev_res);

    return 0;
}