#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h> /* srand, rand */
#include <time.h> /* time */

#include<iostream>
using namespace std;

__host__ void checkCUDAError(const char* msg) {
	hipError_t error;
	hipDeviceSynchronize(); 
	error = hipGetLastError(); 
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), msg);
	}
}

__global__ void idKernel(int* vecA, int* vecB, int* vecC) {
	int gId = threadIdx.x + blockDim.x * blockIdx.x;

	vecA[gId] = threadIdx.x;
	vecB[gId] = blockIdx.x;
	vecC[gId] = gId;
}

void printArray(int* arr, int size, char* msg) {
	cout << msg << ": ";
	for (int i = 0; i < size; i++) {
		printf("%d ", arr[i]);
	}
	printf("\n");
}

int main()
{
	const int vectorSize = 64;
	int* host_a = (int*)malloc(sizeof(int) * vectorSize);
	int* host_b = (int*)malloc(sizeof(int) * vectorSize);
	int* host_c = (int*)malloc(sizeof(int) * vectorSize);

	int* dev_a, * dev_b, * dev_c;

	hipMalloc((void**)&dev_a, sizeof(int) * vectorSize);
	checkCUDAError("Error at hipMalloc for dev_a");
	hipMalloc((void**)&dev_b, sizeof(int) * vectorSize);
	checkCUDAError("Error at hipMalloc for dev_b");
	hipMalloc((void**)&dev_c, sizeof(int) * vectorSize);
	checkCUDAError("Error at hipMalloc for dev_c");

	srand(time(NULL));

	for (int i = 0; i < vectorSize; i++) {
		host_a[i] = 0;
		host_b[i] = 0;
		host_c[i] = 0;
	}

	hipMemcpy(dev_a, host_a, sizeof(int) * vectorSize, hipMemcpyHostToDevice);
	checkCUDAError("Error at hipMemcpy for host_a to dev_a");
	hipMemcpy(dev_b, host_b, sizeof(int) * vectorSize, hipMemcpyHostToDevice);
	checkCUDAError("Error at hipMemcpy for host_b to dev_b");
	hipMemcpy(host_c, dev_c, sizeof(int) * vectorSize, hipMemcpyHostToDevice); // error 1
	checkCUDAError("Error at hipMemcpy for host_c to dev_c");

	dim3 grid(1, 1, 1);
	dim3 block(2000, 1, 1); // max num is 1024, so here we will force an error
	idKernel << < grid, block >> > (dev_a, dev_b, dev_c);
	checkCUDAError("Error at idKernel execution no. 1");
	hipDeviceSynchronize(); // wait until kernel finishes and then come back to following code // not needed to check
	hipMemcpy(host_a, dev_a, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);
	//check also here
	hipMemcpy(host_b, dev_b, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);
	hipMemcpy(host_c, dev_c, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);

	printf("Execution 1: 1 block 64 threads \n");
	printArray(host_a, vectorSize, "threadIdx.x");
	printArray(host_b, vectorSize, "blockIdx.x");
	printArray(host_c, vectorSize, "globalId");

	grid.x = 64; // (64, 1, 1)
	block.x = 1; // (1, 1, 1)
	idKernel << < grid, block >> > (dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	hipMemcpy(host_a, dev_a, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);
	hipMemcpy(host_b, dev_b, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);
	hipMemcpy(host_c, dev_c, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);

	printf("\nExecution 2: 64 blocks 1 thread \n");
	printArray(host_a, vectorSize, "threadIdx.x");
	printArray(host_b, vectorSize, "blockIdx.x");
	printArray(host_c, vectorSize, "globalId");

	grid.x = 4;
	block.x = 16;
	idKernel << < grid, block >> > (dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	hipMemcpy(host_a, dev_a, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);
	hipMemcpy(host_b, dev_b, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);
	hipMemcpy(host_c, dev_c, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);

	printf("\nExecution 3: 4 block 16 threads \n");
	printArray(host_a, vectorSize, "threadIdx.x");
	printArray(host_b, vectorSize, "blockIdx.x");
	printArray(host_c, vectorSize, "globalId");

	free(host_a);
	free(host_b);
	free(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}
