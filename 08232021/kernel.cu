#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void printThreadIds()
{
    int globalId = threadIdx.x + blockDim.x * blockIdx.x;
    printf("globalId: %d, threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d, blockDim.x: %d, blockIdx.x %d\n", globalId, threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockIdx.x);
}

__global__ void printGlobalId_oneBlockOneDim()
{
    printf("GlobalId: %d\n", threadIdx.x);
}

__global__ void printGlobalId_NBlocksOneDim()
{
    int globalId = threadIdx.x + blockDim.x * blockIdx.x;
    printf("GlobalId: %d\n", globalId);
}

int main() {
    dim3 grid(3, 1, 1); 
    dim3 block(4, 1, 1);
    printThreadIds<< < grid, block >> > ();

    return 0;
}
